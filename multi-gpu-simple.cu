
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

int main()
{
    int *a, *b, *c;
    int *d_a1, *d_b1, *d_c1;
    int *d_a2, *d_b2, *d_c2;
    int N = 1000;

    a = (int *)malloc(N * sizeof(int));
    b = (int *)malloc(N * sizeof(int));
    c = (int *)malloc(N * sizeof(int));
    for(int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i;
    }
    hipSetDevice(0);
    hipMalloc(&d_a1, N * sizeof(int));
    hipMalloc(&d_b1, N * sizeof(int));
    hipMalloc(&d_c1, N * sizeof(int));
    hipMemcpy(d_a1, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b1, b, N * sizeof(int), hipMemcpyHostToDevice);
    hipSetDevice(1);
    hipMalloc(&d_a2, N * sizeof(int));
    hipMalloc(&d_b2, N * sizeof(int));
    hipMalloc(&d_c2, N * sizeof(int));
    hipMemcpy(d_a2, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b2, b, N * sizeof(int), hipMemcpyHostToDevice);

    hipSetDevice(0);
    add<<<(N + 255) / 256, 256>>>(d_a1, d_b1, d_c1, N);
    hipSetDevice(1);
    add<<<(N + 255) / 256, 256>>>(d_a2, d_b2, d_c2, N);
    
    hipMemcpy(c, d_c1, N * sizeof(int), hipMemcpyDeviceToHost);
    for(int i = 0; i < N; i++) {
        printf("%d ", c[i]);
    }
    printf("\n");
    hipMemcpy(c, d_c2, N * sizeof(int), hipMemcpyDeviceToHost);
    for(int i = 0; i < N; i++) {
        printf("%d ", c[i]);
    }
    printf("\n");

    hipFree(d_a1);
    hipFree(d_b1);
    hipFree(d_c1);
    hipFree(d_a2);
    hipFree(d_b2);
    hipFree(d_c2);
    free(a);
    free(b);
    free(c);
    return 0;
}